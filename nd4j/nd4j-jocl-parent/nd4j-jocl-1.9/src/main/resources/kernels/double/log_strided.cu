#include "hip/hip_runtime.h"
#include "transform.h"

__device__ double op(double d1,double *params) {
        return log(d1);
}
extern "C"
__global__ void log_strided_double(int n,int idx,double *dy,int incy,double *params,double *result) {
       transform(n,idx,dy,incy,params,result);

 }
