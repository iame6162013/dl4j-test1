#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
__device__ double doBinomial(int n, double p, double *randomNumbers,hiprandGenerator_t s) {
  int x = 0;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  for(int i = tid; i < n; i++) {
    if(randomNumbers[i]< p )
      x++;
  }
  return x;
}


extern "C"
__global__ void binomial_double(int len,int n,double *ps,double *randomNumbers,double *result, hiprandGenerator_t s) {
          int tid = threadIdx.x + blockIdx.x * blockDim.x;
          for(int i = tid; i < len; i += blockDim.x * gridDim.x) {
              result[i] = doBinomial(n,ps[i],randomNumbers,s);
          }
}

