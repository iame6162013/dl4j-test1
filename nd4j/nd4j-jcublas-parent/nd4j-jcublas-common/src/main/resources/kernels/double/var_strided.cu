#include "hip/hip_runtime.h"
#include "reduce.h"


__device__ double merge(double f1,double f2,double *extraParams) {
   return f1 + f2;
}

__device__ double update(double old,double opOutput,double *extraParams) {
       double mean = extraParams[2];
       double curr = powf(opOutput - mean,2.0);
       return old + curr;
 }


//an op for the kernel
__device__ double op(double d1,double *extraParams) {
      return d1;

}

//post process result (for things like means etc)
__device__ double postProcess(double reduction,int n,int xOffset,double *dx,int incx,double *extraParams,double *result) {
             double bias = extraParams[1];
            return  (reduction - (powf(bias,2.0) / n)) / (double) (n - 1.0);

}

extern "C"
__global__ void var_strided_double(int n, int xOffset,double *dx,int incx,double *extraParams,double *result) {
              transform(n,xOffset,dx,incx,extraParams,result);

 }


