#include "hip/hip_runtime.h"
#include "reduce.h"

__device__ float merge(float old,float opOutput,float *extraParams) {
       return opOutput + old;
 }

__device__ float update(float old,float opOutput,float *extraParams) {
       return opOutput + old;
 }


/**
 An op on the device
 @param d1 the first operator
 @param d2 the second operator
*/
__device__ float op(float d1,float d2,float *extraParams) {
      return op(d1,extraParams);
}
//an op for the kernel
__device__ float op(float d1,float *extraParams) {
       float mean = extraParams[1];
       float curr = (d1 - mean);
       return  curr;

}

//post process result (for things like means etc)
__device__ float postProcess(float reduction,int n,int xOffset,float *dx,int incx,float *extraParams,float *result) {
          return reduction;
}

extern "C"
__global__ void bias_strided_float(int n, int xOffset,float *dx,int incx,float *extraParams,float *result) {
              transform(n,xOffset,dx,incx,extraParams,result);

 }


