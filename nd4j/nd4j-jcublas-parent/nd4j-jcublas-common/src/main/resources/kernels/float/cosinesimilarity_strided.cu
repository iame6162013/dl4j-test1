#include "hip/hip_runtime.h"
#include "reduce3.h"


__device__ float update(float old,float opOutput,float *extraParams) {
       return old + opOutput;
 }


/**
 An op on the device
 @param d1 the first operator
 @param d2 the second operator
*/
__device__ float op(float d1,float d2,float *extraParams) {
      return d1 * d2;
}


//post process result (for things like means etc)
__device__ float postProcess(float reduction,int n,int xOffset,float *dx,int incx,float *extraParams,float *result) {
            return reduction / extraParams[1] / extraParams[2];
}

extern "C"
__global__ void cosinesimilarity_strided_float(int n, int xOffset,int yOffset,float *dx,float *dy,int incx,int incy,float *extraParams,float *result) {
              transform_pair(n,xOffset,yOffset,dx,dy,incx,incy,extraParams,result);

 }


