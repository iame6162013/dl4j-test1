extern "C"

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

__global__ void mul_float(int n, float *a, float *b, float *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        sum[i] = a[i] * b[i];
    }

}