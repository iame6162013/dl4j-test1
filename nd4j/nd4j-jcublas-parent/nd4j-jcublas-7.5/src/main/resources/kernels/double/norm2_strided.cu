#include "hip/hip_runtime.h"
#include "reduce.h"

__device__ double merge(double old,double opOutput,double *extraParams) {
       return opOutput + old;
 }

__device__ double update(double old,double opOutput,double *extraParams) {
       return opOutput + old;
 }


__device__ double op(double d1,double *extraParams) {
      return pow(d1,2);
}


__device__ double postProcess(double reduction,int n,int xOffset,double *dx,int incx,double *params,double *result) {
             return sqrtf(reduction);
}
extern "C"
__global__ void norm2_strided_double(int n, int xOffset,double *dx,int incx,double *params,double *result) {
             transform(n,xOffset,dx,incx,params,result);
}


