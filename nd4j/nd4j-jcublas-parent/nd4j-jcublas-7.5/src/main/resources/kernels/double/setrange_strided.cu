#include "hip/hip_runtime.h"
#include "transform.h"


__device__ double op(double d1,double *params) {
       double min = params[0];
       double max = params[1];
    if(d1 >= min && d1 <= max)
               return d1;
           if(min == 0 && max == 1) {
               double val = 1 / (1 + exp(-d1));
               return (floor(val * (max - min)) + min);
           }

           double ret =  (floor(d1 * (max - min)) + min);
           return ret;
     
}

extern "C"
__global__ void setrange_strided_double(int n,int idx,double *dy,int incy,double *params,double *result) {
       transform(n,idx,dy,incy,params,result);

 }
