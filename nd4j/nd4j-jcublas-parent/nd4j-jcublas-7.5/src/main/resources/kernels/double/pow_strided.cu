#include "hip/hip_runtime.h"
#include "transform.h"


__device__ double op(double d1,double *params) {
        return pow(d1,params[0]);
}

extern "C"
__global__ void pow_strided_double(int n,int idx,double *dy,int incy,double *params,double *result) {
       transform(n,idx,dy,incy,params,result);

 }