#include "hip/hip_runtime.h"
#include "scalar.h"
//scalar and current element
__device__ double op(double d1,double d2,double *params) {
   return d1 / d2;
}

extern "C"
__global__ void rdiv_scalar_double(int n, int idx,double dx,double *dy,int incy,double *params,double *result) {
       transform(n,idx,dx,dy,incy,params,result);
 }


