#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
__device__ float doBinomial(int n, float p, float *randomNumbers,hiprandGenerator_t s) {
  int x = 0;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  for(int i = tid; i < n; i++) {
    if(randomNumbers[i]< p )
      x++;
  }
  return x;
}


extern "C"
__global__ void binomial_float(int len,int n,float *ps,float *randomNumbers,float *result, hiprandGenerator_t s) {
          int tid = threadIdx.x + blockIdx.x * blockDim.x;
          for(int i = tid; i < len; i += blockDim.x * gridDim.x) {
              result[i] = doBinomial(n,ps[i],randomNumbers,s);
          }
}

