#include "hip/hip_runtime.h"
#include "reduce3.h"

__device__ float merge(float old,float opOutput,float *extraParams) {
       return old + opOutput;
 }

__device__ float update(float old,float opOutput,float *extraParams) {
       return old + opOutput;
 }


/**
 An op on the device
 @param d1 the first operator
 @param d2 the second operator
*/
__device__ float op(float d1,float d2,float *extraParams) {
      return powf(d1 - d2,2.0f);
}


//post process result (for things like means etc)
__device__ float postProcess(float reduction,int n,int xOffset,float *dx,int incx,float *extraParams,float *result) {
            return sqrt(reduction);
}

extern "C"
__global__ void euclidean_strided_float(int n, int xOffset,int yOffset,float *dx,float *dy,int incx,int incy,float *extraParams,float *result) {
              transform_pair(n,xOffset,yOffset,dx,dy,incx,incy,extraParams,result);

 }


