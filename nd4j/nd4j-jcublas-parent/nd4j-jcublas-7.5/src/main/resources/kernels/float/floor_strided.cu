#include "hip/hip_runtime.h"
#include "transform.h"


__device__ float op(float d1,float *params) {
        return floorf(d1);
}

extern "C"
__global__ void floor_strided_float(int n,int idx,float *dy,int incy,float *params,float *result) {
       transform(n,idx,dy,incy,params,result);

 }
