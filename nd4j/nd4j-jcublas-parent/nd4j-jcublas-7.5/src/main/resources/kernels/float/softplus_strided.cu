#include "hip/hip_runtime.h"
#include "transform.h"


__device__ float op(float d1,float *params) {
          return logf(1 + expf(d1));
}

extern "C"
__global__ void softplus_strided_float(int n,int idx,float *dy,int incy,float *params,float *result) {
       transform(n,idx,dy,incy,params,result);

 }