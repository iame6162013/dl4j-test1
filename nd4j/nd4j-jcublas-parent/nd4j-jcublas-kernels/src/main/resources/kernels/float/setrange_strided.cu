#include "hip/hip_runtime.h"
#include "transform.h"


__device__ float op(float d1,float *params) {
       float min = params[0];
       float max = params[1];
    if(d1 >= min && d1 <= max)
               return d1;
           if(min == 0 && max == 1) {
               float val = 1 / (1 + expf(-d1));
               return (floorf(val * (max - min)) + min);
           }

           float ret =  (floorf(d1 * (max - min)) + min);
           return ret;
     
}

extern "C"
__global__ void setrange_strided_float(int n,int idx,float *dy,int incy,float *params,float *result) {
       transform(n,idx,dy,incy,params,result);

 }
