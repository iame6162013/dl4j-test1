#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include<hiprand/hiprand_kernel.h>


extern "C"
__global__ void normal_float(int n,float *means,float std,float *result, hiprandGenerator_t s) {
        int tid = threadIdx.x + blockIdx.x * blockDim.x;
         for(int i = tid; i < n; i += blockDim.x*gridDim.x) {
                     float *curr = &result[i];
                     hiprandGenerateNormal(s,curr,1,means[i],std);
          }
}

