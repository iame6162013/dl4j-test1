#include "hip/hip_runtime.h"
#include "pairwise_transform.h"

__device__ float op(float d1,float d2,float *params) {
   if(d1 < d2) return 1;
    else return 0;
 }
__device__ float op(float d1,float *params) {
   return d1;
}

extern "C"
__global__ void lt_strided_float(int n,int xOffset,int yOffset, float *dx, float *dy,int incx,int incy,float *params,float *result,int incz) {
    transform(n,xOffset,yOffset,dx,dy,incx,incy,params,result,incz);
}